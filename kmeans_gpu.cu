#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include<iostream>
#include<limits.h>
#include<stdlib.h>
#include<stdio.h>
using namespace std;

int n,c,d;

__global__ void kmeans_gpu(float *point,float *cluster,int *member ,int *d_n,int *d_c,int *d_d )
{
	int change;
	int i,j,k;
	float dist,p; 
	int n=*d_n,c=*d_c,d=*d_d;
	float *newCluster=(float*)(malloc)(sizeof(float)*c*d);
	int *newClusterSize=(int*)(malloc)(sizeof(int)*c);
	for(i=0;i<c;i++)
	{
		for(j=0;j<d;j++)
			newCluster[i*d+j]=0;
		newClusterSize[i]=0;
	}	
	do
	{
		change=0;
		for(i=0;i<n;i++)
		{
			float min_dist=INT_MAX;
			int pos=0;
			for(j=0;j<c;j++)
			{
				dist=0;
				for(k=0;k<d;k++)
				{
					p=(point[i*d+k]-cluster[j*d+k]);
					dist=dist+p*p;
				}
				if(dist<min_dist)
				{
					min_dist=dist;
					pos=j;
				}
			}
			if(member[i]!=pos){
				member[i]=pos;
				change=change+1;
			}
			newClusterSize[pos]++;
			for(k=0;k<d;k++)
			{
				newCluster[pos*d+k]+=point[i*d+k];
			}
		}
		for(i=0;i<c;i++)
		{
			for(k=0;k<d;k++)
			{
				cluster[i*d+k]=newCluster[i*d+k]/newClusterSize[i];
				newCluster[i*d+k]=0;
			}
			newClusterSize[i]=0;
		}
	}while(change >0);
}


void print_clusters(float * cluster)
{
	int i,j;
	for(i=0;i<c;i++)
	{
		for(j=0;j<d;j++)
			printf("%f ",cluster[i*d+j]);
		printf("\n");
	}	

}

void kmeans_gpu_helper(float *point,float *cluster,int *member)
{
	float *d_point;
	float *d_cluster;
	int *d_member;
	int *d_n;
	int *d_c;
	int *d_d;
	
	hipMalloc((void**)&d_point, n*d * sizeof(float));
	hipMalloc((void**)&d_cluster, c*d * sizeof(float));
	hipMalloc((void**)&d_member, c*sizeof(int));
	hipMalloc((void**)&d_n, sizeof(int));
	hipMalloc((void**)&d_c, sizeof(int));
	hipMalloc((void**)&d_d, sizeof(int));
	
	hipMemcpy(d_point, point, n*d * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_cluster, cluster, c*d * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_member, member, d * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n,  sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, &c,  sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_d, &d,  sizeof(int), hipMemcpyHostToDevice);
	
	const dim3 blockSize(1,1,1);
    	const dim3 gridSize(1,1,1);
    	kmeans_gpu<<<gridSize,blockSize>>>(d_point, d_cluster,d_member,d_n,d_c,d_d);
    	hipMemcpy(cluster, d_cluster, c*d * sizeof(float), hipMemcpyDeviceToHost);
}
int main()
{
	/*
	  n-no of points
	  c- no of clusters
	  d- dimensionality of each point 
	 */
	scanf("%d%d%d",&n,&c,&d);
	
	/* Input the co-ordinates */
	
	float * point =(float*)(malloc)(sizeof(float) *n*d);
	float * cluster=(float*)(malloc)(sizeof(float)*c*d);
	int * member=(int*)(malloc)(sizeof(int)*n);
	
	
	int i,j;
	for(i=0;i<n;i++)
	{
		for(j=0;j<d;j++)
		{
			scanf("%f",&point[i*d+j]);
			
		}
		member[i]=0;
	}
	/*Random initialisation using the first k points */
	for(i=0;i<c;i++)
	{
		for(j=0;j<d;j++)
		{
			cluster[i*d+j]=point[i*d+j];
		}
		
	}
	
	kmeans_gpu_helper(point,cluster,member);
	
	print_clusters(cluster);
}
